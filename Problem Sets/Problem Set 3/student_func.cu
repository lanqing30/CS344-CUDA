#include "hip/hip_runtime.h"
/* Udacity Homework 3
   HDR Tone-mapping

  Background HDR
  ==============

  A High Dynamic Range (HDR) image contains a wider variation of intensity
  and color than is allowed by the RGB format with 1 byte per channel that we
  have used in the previous assignment.

  To store this extra information we use single precision floating point for
  each channel.  This allows for an extremely wide range of intensity values.

  In the image for this assignment, the inside of church with light coming in
  through stained glass windows, the raw input floating point values for the
  channels range from 0 to 275.  But the mean is .41 and 98% of the values are
  less than 3!  This means that certain areas (the windows) are extremely bright
  compared to everywhere else.  If we linearly map this [0-275] range into the
  [0-255] range that we have been using then most values will be mapped to zero!
  The only thing we will be able to see are the very brightest areas - the
  windows - everything else will appear pitch black.

  The problem is that although we have cameras capable of recording the wide
  range of intensity that exists in the real world our monitors are not capable
  of displaying them.  Our eyes are also quite capable of observing a much wider
  range of intensities than our image formats / monitors are capable of
  displaying.

  Tone-mapping is a process that transforms the intensities in the image so that
  the brightest values aren't nearly so far away from the mean.  That way when
  we transform the values into [0-255] we can actually see the entire image.
  There are many ways to perform this process and it is as much an art as a
  science - there is no single "right" answer.  In this homework we will
  implement one possible technique.

  Background Chrominance-Luminance
  ================================

  The RGB space that we have been using to represent images can be thought of as
  one possible set of axes spanning a three dimensional space of color.  We
  sometimes choose other axes to represent this space because they make certain
  operations more convenient.

  Another possible way of representing a color image is to separate the color
  information (chromaticity) from the brightness information.  There are
  multiple different methods for doing this - a common one during the analog
  television days was known as Chrominance-Luminance or YUV.

  We choose to represent the image in this way so that we can remap only the
  intensity channel and then recombine the new intensity values with the color
  information to form the final image.

  Old TV signals used to be transmitted in this way so that black & white
  televisions could display the luminance channel while color televisions would
  display all three of the channels.


  Tone-mapping
  ============

  In this assignment we are going to transform the luminance channel (actually
  the log of the luminance, but this is unimportant for the parts of the
  algorithm that you will be implementing) by compressing its range to [0, 1].
  To do this we need the cumulative distribution of the luminance values.

  Example
  -------

  input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
  min / max / range: 0 / 9 / 9

  histo with 3 bins: [4 7 3]

  cdf : [4 11 14]


  Your task is to calculate this cumulative distribution by following these
  steps.

*/

#include "utils.h"

__global__ void reduce_max_min(const float* const d_in, float* d_out, bool is_max=true)
{
	extern __shared__ float partial[];

	int tid = threadIdx.x;
	int idx = blockIdx.x * blockDim.x + tid;

	partial[tid] = d_in[idx];
	// make sure all data in this block has loaded into shared memory
	__syncthreads();

	for(unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1){
		if(tid < stride){
			if(is_max)
				partial[tid] = max(partial[tid], partial[tid+stride]);
			else
				partial[tid] = min(partial[tid], partial[tid+stride]);
		}
		// make sure all operations at one stage are done!
		__syncthreads();
	}


	if(tid == 0)
		d_out[blockIdx.x] = partial[tid];
}

void reduce(const float* const d_in,float &min_logLum,float &max_logLum,const size_t numRows,const size_t numCols)
{

	const int BLOCK_SIZE = numCols;
	const int GRID_SIZE  = numRows;
		// declare GPU memory pointers
	float * d_intermediate, *d_max, *d_min;

	// allocate GPU memory
	hipMalloc((void **) &d_intermediate, GRID_SIZE*sizeof(float));
	hipMalloc((void **) &d_max, sizeof(float));
	hipMalloc((void **) &d_min, sizeof(float));

	// find maximum;
	// firstly, find the maximum in each block
	reduce_max_min<<<GRID_SIZE, BLOCK_SIZE, BLOCK_SIZE*sizeof(float)>>>(d_in, d_intermediate, true);
	// then, find the global maximum
	reduce_max_min<<<1, GRID_SIZE, GRID_SIZE*sizeof(float)>>>(d_intermediate, d_max, true);

	checkCudaErrors(hipMemset(d_intermediate,0,GRID_SIZE*sizeof(float)));
	// find minimum;
	// firstly, find the minimum in each block
	reduce_max_min<<<GRID_SIZE, BLOCK_SIZE, BLOCK_SIZE*sizeof(float)>>>(d_in, d_intermediate, false);
	// then, find the global minimum
	reduce_max_min<<<1, GRID_SIZE, GRID_SIZE*sizeof(float)>>>(d_intermediate, d_min, false);

	// transfer the output to CPU
	checkCudaErrors(hipMemcpy(&max_logLum, d_max, sizeof(float), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(&min_logLum, d_min, sizeof(float), hipMemcpyDeviceToHost));

	// free GPU memory location
	checkCudaErrors(hipFree(d_intermediate));
	checkCudaErrors(hipFree(d_max));
	checkCudaErrors(hipFree(d_min));

	return;
}


__global__ void hist(const float* const d_in, unsigned int * const d_out, const float logLumRange, const int min_logLum, const int numBins)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float num = d_in[idx];
	int bin_idx = (num - min_logLum)/logLumRange*numBins;
	if(bin_idx >= numBins)
		bin_idx--;
	atomicAdd(&(d_out[bin_idx]),1);

}


__global__ void prefixSum_HS(const unsigned int * const d_in, unsigned int * const d_out)
{
	/*	Hillis Steele Scan
		for d := 1 to log2n do
			for all k in parallel do
		 		if k ≥ 2^d then
					x[out][k] := x[in][k − 2^d-1] + x[in][k]
		 		else
					x[out][k] := x[in][k]
		 	swap(in, out)
		This version can handle arrays only as large as can be processed by a single thread block running
		on one multiprocessor of a GPU
	*/
	extern __shared__ unsigned int temp[];

	int tid = threadIdx.x;
	int pout = 0, pin = 1;

	// exclusicve scan
	temp[tid] =  tid > 0? d_in[tid-1]:0;
	// make sure all data in this block are loaded into shared shared memory
	__syncthreads();

	for(unsigned int stride = 1; stride < blockDim.x; stride <<= 1){
		// swap double buffer indices
		pout = 1 - pout;
		pin  = 1 - pout;

		if(tid >= stride)
			temp[pout*blockDim.x+tid] = temp[pin*blockDim.x+tid] + temp[pin*blockDim.x+tid - stride];
		else
			temp[pout*blockDim.x+tid] = temp[pin*blockDim.x+tid];
		// make sure all operations at one stage are done!
		__syncthreads();
	}

	d_out[tid] = temp[pout*blockDim.x + tid];
}


__global__ void prefixSum_BL(const unsigned int * const d_in, unsigned int * const d_out, const int nums)
{
	/* Blelloch Scan : Up-Sweep(reduce) + Down-Sweep
		Up-Sweep:
		for d := 0 to log2n - 1 do
			for k from 0 to n – 1 by 2^(d+1) in parallel do
				x[k + 2^(d + 1) - 1] := x[k + 2^d - 1] + x [k + 2^(d+1) - 1]

		Down-Sweep:
		x[n - 1] := 0
		for d := log2n down to 0 do
			for k from 0 to n – 1 by 2^(d+1) in parallel do
				t := x[k + 2^d- 1]
				x[k + 2^d - 1] := x [k + 2^(d+1) - 1]
				x[k + 2^(d+1) - 1] := t + x [k + 2^(d+1) - 1]
	*/
	extern __shared__ unsigned int temp[];

	int tid = threadIdx.x;
	// exclusicve scan

	temp[2*tid] = d_in[2*tid];
	if(2*tid+1 < nums)
		temp[2*tid+1] = d_in[2*tid+1];
	else
		temp[2*tid+1] = 0;

	// make sure all data in this block are loaded into shared memory
	__syncthreads();

	int stride = 1;
	// reduce step
	for(unsigned int d = blockDim.x; d > 0; d >>= 1){
		if(tid < d) {
			int idx1 = (2*tid+1)*stride - 1;
			int idx2 = (2*tid+2)*stride - 1;
			temp[idx2] += temp[idx1];
		}
		stride *= 2;
		// make sure all operations at one stage are done!
		__syncthreads();
	}

	// Downsweep Step
	// set identity value
	if(tid == 0)
		temp[nums-1] = 0;
	for(unsigned int d = 1; d < nums; d <<= 1){
		stride >>= 1;
		// make sure all operations at one stage are done!
		__syncthreads();
		if( tid < d){
			int idx1 = (2*tid+1)*stride - 1;
			int idx2 = (2*tid+2)*stride - 1;
			unsigned int tmp  = temp[idx1];
			temp[idx1] = temp[idx2];
			temp[idx2] += tmp;
		}
	}
	// make sure all operations at the last  stage are done!
	__syncthreads();
	d_out[2*tid] = temp[2*tid];
	if(2*tid+1 < nums)
		d_out[2*tid+1] = temp[2*tid+1];
}

// Scan algorithm from Course : Hetergeneous Parallel Programming
__global__ void prefixSum_HPP(const unsigned int * const d_in, unsigned int * const d_out, const int nums)
{

	extern __shared__ unsigned int temp[];

	int tid = threadIdx.x;

	// exclusicve scan
	if(tid == 0){
		temp[2*tid] = 0;
		temp[2*tid+1] = d_in[2*tid];
	}
	else{
		temp[2*tid] = d_in[2*tid-1];
		if(2*tid+1 < nums)
			temp[2*tid+1] = d_in[2*tid];
		else
			temp[2*tid+1] = 0;
	}
	// make sure all data in this block are loaded into shared shared memory
	__syncthreads();

	// Reduction Phase
	for(unsigned int stride = 1; stride <= blockDim.x; stride <<= 1){
		// first update all idx == 2n-1, then 4n-1, then 8n-1 ...
		// finaly 2(blockDim.x/2) * n - 1(only 1 value will be updated partial[blockDim.x-1])
		int idx = (tid+1)*stride*2 - 1;
		if( idx  < 2*blockDim.x)
			temp[idx] += temp[idx-stride];
		// make sure all operations at one stage are done!
		__syncthreads();
	}
	// Example:
	// After reduction phase , position at 0, 1, 3, 7, ... has their final values (blockDim.x == 8)
	// then we update values reversely.
	// first use position 3's value to update position 5(stride == 2 == blockDim.x/4, idx == 3 == (0+1)*2*2-1, only 1 thread do calculation)
	// then use position 1 to update postion 2 , position 3 to update position 4, position 5 to update position 6
	//			(stride == 1 == blockDim.x/8, idx == (0+1)*1*2-1=1,(1+1)*1*2-1=3, (2+1)*1*2-1=5, 3 threads do calculation)

	// Post Reduction Reverse Phase
	for(unsigned int stride = blockDim.x/2; stride > 0; stride >>= 1){
		// first update all idx == 2(blockDim.x/4) * n - 1 + blockDim.x/4,
		// then 2(blockDim.x/8)n-1+blockDim.x/8, then 2(blockDim.x/16)n-1 + blockDim.x/16...
		// finaly 2 * n - 1
		int idx = (tid+1)*stride*2 - 1;
		if( idx + stride  < 2*blockDim.x)
			temp[idx + stride] += temp[idx];
		// make sure all operations at one stage are done!
		__syncthreads();
	}

	// exclusive scan

	d_out[2*tid] = temp[2*tid];
	if(2*tid+1 < nums)
		d_out[2*tid+1] = temp[2*tid+1];
}

void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
  //TODO
  /*Here are the steps you need to implement
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum
    2) subtract them to find the range
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */


	// Step 1 : find minimum and maximum value
	reduce(d_logLuminance, min_logLum, max_logLum, numRows, numCols);


	// Step 2: find the range
	float logLumRange = max_logLum - min_logLum;

	// Step 3 : generate a histogram of all the values
	// declare GPU memory pointers
	unsigned int  *d_bins;
	// allocate GPU memory
	checkCudaErrors(hipMalloc((void **) &d_bins, numBins*sizeof(unsigned int)));
	checkCudaErrors(hipMemset(d_bins,0,numBins*sizeof(unsigned int)));

	hist<<<numRows, numCols>>>(d_logLuminance, d_bins, logLumRange, min_logLum, numBins);

	// Step 4 : prefix sum
	//prefixSum_HS<<<1, numBins, numBins*sizeof(unsigned int)>>>(d_bins, d_cdf);
	//prefixSum_HPP<<<1, ceil(numBins/2), numBins*sizeof(unsigned int)>>>(d_bins, d_cdf, numBins);
	prefixSum_BL<<<1, ceil(numBins/2), numBins*sizeof(unsigned int)>>>(d_bins, d_cdf, numBins);
	// free GPU memory allocation
	checkCudaErrors(hipFree(d_bins));
}
